
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
using namespace std;

struct blockm
{
    int i;
    int j;
    vector<short>block_value;
};

struct cords
{
  int i;
  int j;
};

struct outblocks
{
  int i;
  int j;
  vector<int>data;
};

__global__ void matrixMul(const short *a, const short *b, int *c,int n) {
  
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  long long temp = (long long)0;

  if(row< n && col < n)
  {   
      for(int i=0;i<n;i++)
      {
          temp+=(long long )a[row*n+i]*(long long )b[i*n+col];
      }

      if(temp>4294967295)
             temp=4294967295;

      c[row*n+col]=(unsigned int)temp;       
               
  }

}


void linearize(vector<vector<short>>&fmatrix,vector<blockm>&tblocks,int nz,int m)
{
 for(int i=0;i<nz;i++)
 {
    blockm temp=tblocks[i];

    int sri=temp.i;
    int csi=temp.j;
    int k=0;
    
    for(int g=sri*m;g<sri*m+m;g++)
    {
        for(int h=csi*m;h<csi*m+m;h++)
            {
                fmatrix[g][h]=temp.block_value[k];
                k++;
            }
    }

  }

}

void naive(short *p1,int *o,int n)
{
  
    for (int i = 0; i < n; i++) {
        for (int j = 0; j <n; j++) {
            //  rslt[i][j] = 0;
             o[i*n+j]=0;
  
            for (int k = 0; k < n; k++) {
                cout<<"value is following :: "<<p1[i*k+j]<<endl;
                o[i*n+j] += (int)p1[i*n+k] * (int)p1[i*k+j];
            }
        }
  
    }
}


int main(int argc, char* argv[])
{
  string fn1(argv[1]);
  string fn2(argv[2]);
  string ofn(argv[3]);

  // cout<<"f1 is "<<fn2<<endl;


int n,m;
int nz;
int nz2;

ifstream infile;
ifstream infile2;

infile.open(fn1,ios::binary);
infile2.open(fn2,ios::binary);

infile.read((char*)&n,4);
infile.read((char*)&m,4);
infile.read((char*)&nz,4);

infile2.read((char*)&n,4);
infile2.read((char*)&m,4);
infile2.read((char*)&nz2,4);


vector<blockm>tblocks;

for(int i=0;i<nz;i++)
{
    blockm temp;
    infile.read((char*)&temp.i,4);
    infile.read((char*)&temp.j,4);   

    for(int i=0;i<m*m;i++)
    {
        short value;
        infile.read((char*)&value,2);
        temp.block_value.push_back(value);
    }
    tblocks.push_back(temp);
}

vector<vector<short>>fmatrix(n,vector<short>(n,(short)0));
linearize(fmatrix,tblocks,nz,m);
tblocks.clear();


//taking input2::
for(int i=0;i<nz2;i++)
{
    blockm temp;
    infile2.read((char*)&temp.i,4);
    infile2.read((char*)&temp.j,4);   

    for(int i=0;i<m*m;i++)
    {
        short value;
        infile2.read((char*)&value,2);
        temp.block_value.push_back(value);
    }
    tblocks.push_back(temp);
}

vector<vector<short>>fmatrix2(n,vector<short>(n,(short)0));
linearize(fmatrix2,tblocks,nz2,m);
tblocks.clear();


short * a=(short*)malloc(n*n*sizeof(short));
short * b=(short*)malloc(n*n*sizeof(short));

int counter=0;
for(int i=0;i<fmatrix.size();i++)
{
    for(int j=0;j<fmatrix[0].size();j++)
    {
        a[counter]=fmatrix[i][j];
        b[counter]=fmatrix2[i][j];
        counter++;
    }
}

fmatrix.clear();
fmatrix2.clear();



short* input1;
short* input2;
int * out;

size_t bytes1 = n*n*sizeof(short);
size_t bytes2 = n*n*sizeof(int);

hipMalloc(&input1,bytes1);
hipMalloc(&input2,bytes1);
hipMalloc(&out,bytes2);

 // Copy data to the device
  hipMemcpy(input1, a,n*n*sizeof(short), hipMemcpyHostToDevice);
  hipMemcpy(input2, b,n*n*sizeof(short), hipMemcpyHostToDevice);
 

  int BLOCK_SIZE = 32;
  int GRID_SIZE=(int)ceil((float)n/BLOCK_SIZE);

  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 blocks(GRID_SIZE,GRID_SIZE);

  matrixMul<<<blocks, threads>>>(input1,input2,out,n);

  
  int * output=(int*)malloc(n*n*sizeof(int));
  hipError_t err=hipMemcpy(output, out,n*n*sizeof(int), hipMemcpyDeviceToHost);  
  

  hipFree(input1);
  hipFree(input2);
  hipFree(out);

//************************************DONE MULTIPLYING*************************************************************************************
  vector<vector<int>>matrix;
  vector<int>block11;

    for(int i=0;i<n*n;i++)
    { 
      if((i+1)%n==0)
       {
        // cout<<output[i];
        
        block11.push_back(output[i]);
        matrix.push_back(block11);
        block11.clear();
        
        // cout<<endl;
       }
    else 
      {
        // cout<<output[i]<<" ";
       
       block11.push_back(output[i]);
      }
    }

  free(output);
  

  ofstream outfile(ofn,ios::binary);
  outfile.write((char *)&n,sizeof(int));
  outfile.write((char *)&m,sizeof(int));


  vector<outblocks>output333;
    
    int block_size=m;

    for (int i = 0; i < matrix.size(); i += block_size) {
        for (int j = 0; j < matrix.size(); j += block_size) {
            vector<int> block;
            int count=0;
            
            for (int x = i; x < i + block_size; x++) {
                for (int y = j; y < j + block_size; y++) {
                    
                    int data=matrix[x][y];

                    if(data==0)
                          count++;
                    block.push_back(matrix[x][y]);
                }
            }
            int ind=i/m;
            int ind2=j/m;

            if(count==m*m)
                 {
                  continue;
                 }

            else
            {
              outblocks temp111;
              temp111.i=ind;
              temp111.j=ind2;

              for(int i=0;i<block.size();i++)
               {
                      int value=block[i];
                      temp111.data.push_back(value);

               }
              output333.push_back(temp111);
              block.clear();   
            }    
        }
    }


int sz=output333.size();

outfile.write((char *)&sz,sizeof(int));


for(int i=0;i<output333.size();i++)
{
     outfile.write((char *)&output333[i].i,sizeof(int));
     outfile.write((char *)&output333[i].j,sizeof(int));

     for(int  j=0;j<output333[i].data.size();j++)
     {
      int value=output333[i].data[j];
      outfile.write((char *)&value,sizeof(int));
     
     }

}

output333.clear();
outfile.close();
matrix.clear();

return 0;
             
}
